
#include <hip/hip_runtime.h>
__global__ void addVectors(float *a, float *b, float *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

// Wrapper function that will be called from C++
extern "C" void launchAddVectors(float *a, float *b, float *c, int n) {
    float *d_a, *d_b, *d_c;
    
    // Allocate device memory
    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));
    
    // Copy inputs to device
    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    addVectors<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
    
    // Copy result back to host
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
